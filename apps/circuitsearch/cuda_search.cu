#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>

#include <thrust/complex.h>

/* Performance

cuda_search 32 1024 indicas 85ms to decompose the gate (1024x32 threads)

*/


#define GPU 1

typedef thrust::complex<float> cplx;
typedef cplx onebit_op[2][2];
typedef cplx twobit_op[4][4];

#define new_cplx(r,i)  
cplx Id[2][2] = {{1,0}, {0,1}};
cplx X[2][2] = {{0,1}, {1,0}};
cplx Y[2][2] = {{0,cplx(0,-1)}, {cplx(0,1),0}};
cplx Z[2][2] = {{1,0}, {0,-1}}; 
cplx S[2][2] = {{1,0}, {0,cplx(0, 1)}}; 
cplx T[2][2] = {{1,0}, {0, 0}}; 
cplx Td[2][2]; 

cplx CNOT[4][4] = {{1,0,0,0}, {0,1,0,0}, {0,0,0,1}, {0,0,1,0}};

cplx CS[4][4] = {{1,0,0,0}, {0,1,0,0}, {0,0,1,0}, {0,0,0,cplx(0, 1)}};

#define num_1d_ops 7
onebit_op* obops[num_1d_ops] = {&Id, &X, &Y, &Z, &S, &T, &Td};
const char* obops_name[num_1d_ops] = {"Id", "X", "Y", "Z", "S", "T", "Td"};

// Plus 1 for CNOT, don't add CS that is target.
// num_1d_ops*num_1d_ops + 1
#define num_pairs 50

twobit_op *pairs;
char* pairs_name[num_pairs];

// C = A*B as matrices.
void multiply(cplx A[2][2], cplx B[2][2], cplx C[2][2])
{
    cplx c00, c01, c10, c11;
    c00 = A[0][0]*B[0][0] + A[0][1]*B[1][0];
    c01 = A[0][0]*B[0][1] + A[0][1]*B[1][1];
    c10 = A[1][0]*B[0][0] + A[1][1]*B[1][0];
    c11 = A[1][0]*B[0][1] + A[1][1]*B[1][1];

    C[0][0] = c00;
    C[0][1] = c01;
    C[1][0] = c10;
    C[1][1] = c11;
}

// B = k*A, k scalar, B 2-dim matrix .
void scalar_multiply(cplx k, cplx A[2][2], cplx B[2][2])
{
    B[0][0] = k*A[0][0];
    B[0][1] = k*A[0][1];
    B[1][0] = k*A[1][0];
    B[1][1] = k*A[1][1];
}


// C = A tensor B where A, B are 2-dim matrices.
void tensor(cplx A[2][2], cplx B[2][2], cplx C[4][4])
{
    onebit_op tmp;
    scalar_multiply(A[0][0], B, tmp);
    C[0][0] = tmp[0][0];
    C[0][1] = tmp[0][1];
    C[1][0] = tmp[1][0];
    C[1][1] = tmp[1][1];

    scalar_multiply(A[0][1], B, tmp);
    C[0][2] = tmp[0][0];
    C[0][3] = tmp[0][1];
    C[1][2] = tmp[1][0];
    C[1][3] = tmp[1][1];

    scalar_multiply(A[1][0], B, tmp);
    C[2][0] = tmp[0][0];
    C[2][1] = tmp[0][1];
    C[3][0] = tmp[1][0];
    C[3][1] = tmp[1][1];

    scalar_multiply(A[1][1], B, tmp);
    C[2][2] = tmp[0][0];
    C[2][3] = tmp[0][1];
    C[3][2] = tmp[1][0];
    C[3][3] = tmp[1][1];
}


void copy4x4(cplx A[4][4], cplx B[4][4])
{
    int i, j;
    for (i=0; i<4; i++)
    {
        for (j=0; j<4; j++)
        {
            B[i][j] = A[i][j];
        }
    }
}


void print_cplx(cplx &v) {
  printf("%.2f+%.2fi\t", v.real(), v.imag());
}

void print2x2(cplx A[2][2])
{
  print_cplx(A[0][0]);
  print_cplx(A[0][1]);
  printf("\n");
  print_cplx(A[1][0]);
  print_cplx(A[1][1]);
  printf("\n\n");
}

// Create all pairs of one dim operators, then adjoin two bit operators.
void setup_pairs()
{
#ifdef GPU
  hipError_t err = hipMallocManaged(&pairs, num_pairs * 16 * sizeof(cplx));
  if (err != hipSuccess) {
    printf("Failed to allocated memory!\n");
    exit(1);
  }
#else
  pairs = (twobit_op*)calloc(num_pairs, sizeof(twobit_op));
#endif
  int i, j, k;
  char buf[64];
  
  for (i=0; i<num_1d_ops; i++)
    {
      for (j=0; j<num_1d_ops; j++)
        {
  	  k = i*num_1d_ops + j;
  	  tensor(*obops[i], *obops[j], pairs[k]);
	  
  	  // Save name of pair
  	  sprintf(buf, "%s %s", obops_name[i], obops_name[j]);
  	  pairs_name[k] = strdup(buf);
        }
    }
  
  copy4x4(CNOT, pairs[k+1]);
  pairs_name[k+1] = strdup("CNOT");
}

void init() {
  T[1][1] = cplx(1, 1) / sqrt(cplx(2,0));

  // Create Td
  multiply(T, Z, Td);
  multiply(Td, S, Td);
}

// C = A*B as 4x4 matrices.
#ifdef GPU
__device__
#endif
void multiply4x4(twobit_op A, twobit_op B, twobit_op C)
{
    int i, j, k;
    cplx sum;

    for (i=0; i<4; i++)
    {
        for (j=0; j<4; j++)
        {
            sum = 0;
            for (k=0; k<4; k++)
            {
                sum = sum + A[i][k] * B[k][j];
            }
            C[i][j] = sum;
        }
    }
}

#ifdef GPU
__device__
#endif
float cabsf(cplx x)
{
  return sqrt(x.real()*x.real() + x.imag() * x.imag());
}

#ifdef GPU
__device__
#endif
int equals(cplx A[4][4], cplx B[4][4])
{
    int i, j;
    for (i=0; i<4; i++)
    {
        for (j=0; j<4; j++)
        {
            if (cabsf(B[i][j] - A[i][j]) > 0.1)
            {
                return 0;
            }
        }
    }
    return 1;
}

// GPU function to find gate combination
#ifdef GPU
__global__
#endif
void find(twobit_op *pairs, long n_combinations)
{

    cplx target[4][4] = {{1,0,0,0}, {0,1,0,0}, {0,0,1,0}, {0,0,0,cplx(0, 1)}};

    int i1, i2, i3, i4;
    twobit_op tmp1, tmp2, tmp3;
    long k=0;

    int match = 0;
    int num_pairs_2 = num_pairs * num_pairs;
    int num_pairs_3 = num_pairs_2 * num_pairs;
    long i = 0;

#ifdef GPU
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
#else
    int index = 0;
    int stride = 1;
#endif
    for (i = index; i < n_combinations; i += stride) {
      i1 = i % num_pairs;
      i2 = (i / num_pairs) % num_pairs;
      i3 = (i / num_pairs_2) % num_pairs;
      i4 = (i / num_pairs_3) % num_pairs;
      
      multiply4x4(pairs[i1], pairs[i2], tmp1);
      multiply4x4(tmp1, pairs[i3], tmp2);
      multiply4x4(tmp2, pairs[i4], tmp3);
      if (equals(target, tmp3))
      	{
      	  match++;
      	  printf("%d : %d : %d : %d\n", i1, i2, i3, i4);
      	}
      k++;
    }
    /* printf("found %d matches in %ld combinations\n", match, k); */
    /* printf("stopped search after %ld million loops\n", k/1000000); */
}



int main(int argc, char *argv[])
{
#ifdef GPU
  printf("running in GPU mode\n");
#else
  printf("running in non GPU mode\n");
#endif

  int blockSize = atoi(argv[1]);
  int numBlocks = atoi(argv[2]);
    
  init();
  
  setup_pairs();

  int n_gates = 4;
  long n_combinations = (long) pow(num_pairs, n_gates);

#ifdef GPU

  printf("using %d blocks with blockSize %d\n", numBlocks, blockSize);
  find<<<numBlocks, blockSize>>>(pairs, n_combinations);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("failed to synchronize cuda device\n");
    printf("%s\n", hipGetErrorString(err));
  }
#else
  find(pairs, n_combinations);
#endif
  
#ifdef GPU
  hipFree(pairs);
#endif
  
  return 0;
}
